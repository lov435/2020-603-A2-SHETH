#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include <list>
#include <map>
#include "../libarff/arff_parser.h"
#include "../libarff/arff_data.h"

using namespace std;

struct Neighbor {
	float distance;
	int cls;
};

struct Instance {
	float * attribs;
	int cls;
};

int majorityVote(list<Neighbor> & neighbors) {
    std::map<int, int> frequencyMap;
    int maxFrequency = 0;
    int  mostFrequentClass = -1;
	//cout << "Neighbors are " << endl;
    for (Neighbor neighbor : neighbors)
    {
		//cout << neighbor.distance << "  " << neighbor.cls << endl;
        int f = ++frequencyMap[neighbor.cls];
        if (f > maxFrequency)
        {
            maxFrequency = f;
            mostFrequentClass = neighbor.cls;
        }
    }

    return mostFrequentClass;
}

__device__ int majorityVote(int k, Neighbor * neighbors) {

}

__global__ void predictForOneInstance(Instance * instances, int numInstances, int numAttribs,
		int k, int * prediction)
{
	//First, compute the thread id and call it i.
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    Instance current_instance = instances[i];

    //Array of k neighbors. Initialize them.
	Neighbor * neighbors = (Neighbor *)malloc(sizeof(Neighbor)*k);
	for(int p=0; p <k; p++) {
		neighbors[p].distance = FLT_MAX;
		neighbors[p].cls = -1;
	}


	for(int j = 0; j < numInstances; j++) // target each other instance
	{
		if(i == j) continue;

		float distance = 0;

		for(int h = 0; h < numAttribs - 1; h++) // compute the distance between the two instances
		{
			float diff = current_instance.attribs[h] - instances[j].attribs[h];
			distance += diff * diff;
		}

		distance = sqrt(distance);

		for(int p=0; p <k; p++) {
			if(distance < neighbors[p].distance) {
				Neighbor neighbor;
				neighbor.distance = distance;
				neighbor.cls = instances[j].cls;

				Neighbor * newNeighbors = (Neighbor *)malloc(sizeof(Neighbor)*k);

				for(int q=0, r=0; q <k; q++) {
					if(p == q) {
						newNeighbors[q] = neighbor;
						continue;
					}
					newNeighbors[q] = neighbors[r++];
				}

				free(neighbors);
				neighbors = newNeighbors;
				break;
			}
		}
	}
	*prediction = majorityVote(k, neighbors);
	//Free the memory
	free(neighbors);
}

int* KNN(ArffData* dataset, int k)
{
    // predictions is the array where you have to return the class predicted (integer) for the dataset instances
    int* predictions = (int*)malloc(dataset->num_instances() * sizeof(int));
    
    // The following two lines show the syntax to retrieve the attribute values and the class value for a given instance in the dataset
    // float attributeValue = dataset->get_instance(instanceIndex)->get(attributeIndex)->operator float();
    // int classValue =  dataset->get_instance(instanceIndex)->get(dataset->num_attributes() - 1)->operator int32();
    
    // Implement the KNN here, fill the predictions array
	cout << "K is " << k << endl;
	
	for(int i = 0; i < dataset->num_instances(); i++) // for each instance in the dataset
	{
		float smallestDistance = FLT_MAX;
		int smallestDistanceClass;

		//Initialize an empty neighbor
		Neighbor  neighbor;
		neighbor.distance = FLT_MAX;
		neighbor.cls = -1;
		
		//List of k neighbors
		std::list<Neighbor> neighbors (k, neighbor);
	
		for(int j = 0; j < dataset->num_instances(); j++) // target each other instance
		{
			if(i == j) continue;
			
			float distance = 0;
			
			for(int h = 0; h < dataset->num_attributes() - 1; h++) // compute the distance between the two instances
			{
				float diff = dataset->get_instance(i)->get(h)->operator float() - dataset->get_instance(j)->get(h)->operator float();
				distance += diff * diff; 
			}
			
			distance = sqrt(distance);
			
			for (std::list<Neighbor>::iterator it = neighbors.begin(); it != neighbors.end(); it++) {
				if(distance < (*it).distance) {
					Neighbor neighbor;
					neighbor.distance = distance;
					neighbor.cls = dataset->get_instance(j)->get(dataset->num_attributes() - 1)->operator int32();
					neighbors.insert(it, neighbor);
					neighbors.pop_back(); //Remove the last neighbor
					break;
				}
			}
		}
		
		predictions[i] = majorityVote(neighbors);
	}
	
    return predictions;
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset)
{
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses
    
    for(int i = 0; i < dataset->num_instances(); i++) // for each instance compare the true class and predicted class
    {
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];
        
        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }
    
    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset)
{
    int successfulPredictions = 0;
    
    for(int i = 0; i < dataset->num_classes(); i++)
    {
        successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagonal are correct predictions
    }
    
    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char *argv[])
{
    if(argc != 3)
    {
        cout << "Usage: ./main <path to dataset> <k>" << endl;
        exit(0);
    }
    
    // Open the dataset
    ArffParser parser(argv[1]);
    ArffData *dataset = parser.parse();
    struct timespec start, end;
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    
    // Get the class predictions
    int* predictions = KNN(dataset, atoi(argv[2]));
    // Compute the confusion matrix
    int* confusionMatrix = computeConfusionMatrix(predictions, dataset);
    // Calculate the accuracy
    float accuracy = computeAccuracy(confusionMatrix, dataset);
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;
  
    printf("The KNN classifier for %lu instances required %llu ms CPU time, accuracy was %.4f\n", dataset->num_instances(), (long long unsigned int) diff, accuracy);
	
	//Free the memory
	free(predictions);
	free(confusionMatrix);
}
