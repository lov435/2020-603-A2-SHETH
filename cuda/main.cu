#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include <list>
#include <map>
#include "../libarff/arff_parser.h"
#include "../libarff/arff_data.h"

using namespace std;

struct Neighbor {
	float distance;
	int cls;
};

struct Instance {
	float * attribs;
	int cls;
};

int majorityVote(list<Neighbor> & neighbors) {
    std::map<int, int> frequencyMap;
    int maxFrequency = 0;
    int  mostFrequentClass = -1;
	//cout << "Neighbors are " << endl;
    for (Neighbor neighbor : neighbors)
    {
		//cout << neighbor.distance << "  " << neighbor.cls << endl;
        int f = ++frequencyMap[neighbor.cls];
        if (f > maxFrequency)
        {
            maxFrequency = f;
            mostFrequentClass = neighbor.cls;
        }
    }

    return mostFrequentClass;
}

void predictFromDistances (float * distances, Instance * instances, int numInstances,
			int k, int * predictions) {

	//Initialize an empty neighbor
	Neighbor  neighbor;
	neighbor.distance = FLT_MAX;
	neighbor.cls = -1;
	//List of k neighbors
	std::list<Neighbor> neighbors (k, neighbor);

	for(int i=0; i < numInstances * numInstances; i++) {
		if(i%numInstances == i/numInstances)
			distances[i] = FLT_MAX; //subject and target instances are the same.

		for (std::list<Neighbor>::iterator it = neighbors.begin(); it != neighbors.end(); it++) {
			if(distances[i] < (*it).distance) {
				Neighbor neighbor;
				neighbor.distance = distances[i];
				neighbor.cls = instances[i%numInstances].cls;
				neighbors.insert(it, neighbor);
				neighbors.pop_back(); //Remove the last neighbor
				break;
			}
		}

		if((i+1)%numInstances == 0) {
			predictions[i/numInstances] = majorityVote(neighbors);
			//Reset the neighbors as we are starting the next instance
			neighbors.clear();
			for (int x=0; x <k; x++)
			{
				Neighbor neighbor;
				neighbor.distance = FLT_MAX;
				neighbor.cls = -1;
				neighbors.push_back(neighbor);
			}
		}

	}
}

__device__ int majorityVote(int k, Neighbor * neighbors) {
	struct FrequencyMap {
		int cls;
		int freq;
	};

	FrequencyMap * freqMap = (FrequencyMap *)malloc(sizeof(FrequencyMap)*k);

    int maxFrequency = 0;
    int  mostFrequentClass = neighbors[0].cls; //default, useful when k is 1
    int numClasses = 0;

	for(int i=0; i <k; i++) {
		bool found = false;
		for(int j=0; j < numClasses; j++) {
			if(freqMap[j].cls == neighbors[i].cls) {
				found = true;
				freqMap[j].freq = freqMap[j].freq + 1;
				if(freqMap[j].freq > maxFrequency) {
					maxFrequency = freqMap[j].freq;
					mostFrequentClass = freqMap[j].cls;
				}
				break;
			}
		}
		if(!found) {
			//Encountered this class first time. Add it to the map.
			freqMap[numClasses].cls = neighbors[i].cls;
			freqMap[numClasses].freq = 1;
			numClasses++;
		}
	}
	free(freqMap);
	return mostFrequentClass;
}


/*
 * Advance CUDA kernel function.
 * Each thread simply calculates the distance of one specific instance to another specific one.
 * Therefore, this model requires as many threads as the number of elements in the dataset
 */
__global__ void advanceCuda(Instance * instances, int numInstances, int numAttribs, float * distances)
{
	//First, compute the thread id and call it i.
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= numInstances * numInstances) {
    	return;
    }

    Instance current_instance = instances[i/numInstances];
	Instance target_instance = instances[i%numInstances];

	float distance = 0;
	for(int h = 0; h < numAttribs; h++) // compute the distance between the two instances
	{
		float diff = current_instance.attribs[h] - target_instance.attribs[h];
		distance += diff * diff;
	}

	distance = sqrt(distance);

	distances[i] = distance;
}


/*
 * Basic CUDA kernel function.
 * Each threads runs KNN for exactly one instance in the dataset.
 * Therefore, this model requires as many threads as the number of elements in the dataset
 */
__global__ void basicCuda(Instance * instances, int numInstances, int numAttribs,
		int k, int * prediction)
{
	//First, compute the thread id and call it i.
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= numInstances) {
    	return;
    }

    Instance current_instance = instances[i];

    //Array of k neighbors. Initialize them.
	Neighbor * neighbors = (Neighbor *)malloc(sizeof(Neighbor)*k);
	for(int p=0; p <k; p++) {
		neighbors[p].distance = FLT_MAX;
		neighbors[p].cls = -1;
	}


	for(int j = 0; j < numInstances; j++) // target each other instance
	{
		if(i == j) continue;

		float distance = 0;

		for(int h = 0; h < numAttribs; h++) // compute the distance between the two instances
		{
			float diff = current_instance.attribs[h] - instances[j].attribs[h];
			distance += diff * diff;
		}

		distance = sqrt(distance);

		for(int p=0; p <k; p++) {
			if(distance < neighbors[p].distance) {
				Neighbor neighbor;
				neighbor.distance = distance;
				neighbor.cls = instances[j].cls;

				Neighbor * newNeighbors = (Neighbor *)malloc(sizeof(Neighbor)*k);

				for(int q=0, r=0; q <k; q++) {
					if(p == q) {
						newNeighbors[q] = neighbor;
						continue;
					}
					newNeighbors[q] = neighbors[r++];
				}

				free(neighbors);
				neighbors = newNeighbors;
				break;
			}
		}
	}
	prediction[i] = majorityVote(k, neighbors);
	//Free the memory
	free(neighbors);
}

int* advanceCudaKNN(ArffData* dataset, int k)
{
    // predictions is the array where you have to return the class predicted (integer) for the dataset instances
    int* predictions = (int*)malloc(dataset->num_instances() * sizeof(int));

    // The following two lines show the syntax to retrieve the attribute values and the class value for a given instance in the dataset
    // float attributeValue = dataset->get_instance(instanceIndex)->get(attributeIndex)->operator float();
    // int classValue =  dataset->get_instance(instanceIndex)->get(dataset->num_attributes() - 1)->operator int32();

    // Implement the KNN here, fill the predictions array
	cout << "K is " << k << endl;

	int numElements = dataset->num_instances();
	int numAttribs = dataset->num_attributes() - 1; //-1 because the last attrib is class

	Instance * h_instances = (Instance *)malloc(numElements * sizeof(Instance));

	// Launch the CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements * numElements / threadsPerBlock) + 1;

	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	//Convert the arf dataset to an array of Instance structure on host
	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		float * attribs = (float *) malloc (sizeof(float)*numAttribs);
		for(int h = 0; h < numAttribs; h++) // compute the distance between the two instances
		{
			attribs[h] = dataset->get_instance(i)->get(h)->operator float();
		}
		h_instances[i].attribs = attribs;
		h_instances[i].cls = dataset->get_instance(i)->get(numAttribs)->operator int32();
	}

	//Make another copy of the instances array from host to device
    Instance * d_instances;
	hipMalloc(&d_instances, numElements*sizeof(Instance));
	hipMemcpy(d_instances, h_instances, numElements*sizeof(Instance), hipMemcpyHostToDevice);
	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		float * d_attribs;
		hipMalloc(&d_attribs, numAttribs*sizeof(float));
		// Copy up attributes for each instance separately
		hipMemcpy(d_attribs, h_instances[i].attribs, numAttribs*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&(d_instances[i].attribs), &d_attribs, sizeof(float*), hipMemcpyHostToDevice);
	}

	//Create an array of numElements X numElements elements on device. Kernel function will
	//populate the distances here.
	float * d_distances;
	hipMalloc(&d_distances, numElements*numElements*sizeof(float));

	//Call kernel
	advanceCuda<<<blocksPerGrid, threadsPerBlock>>>(d_instances, numElements, numAttribs, d_distances);

    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess) {
        fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

	//Create an array of numElements X numElements elements on host.
	//We will copy the distance array from the device to this one
	float * h_distances = (float *) malloc(numElements*numElements*sizeof(float));
	// Copy the device distance vector in device memory to the host distance vector
    hipMemcpy(h_distances, d_distances, numElements * numElements * sizeof(float), hipMemcpyDeviceToHost);

	predictFromDistances(h_distances, h_instances, numElements, k, predictions);

    // Free host memory
   	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		free(h_instances[i].attribs);
	}
    free(h_instances);
	free(h_distances);

    // Free device global memory
    hipFree(d_distances);
	Instance * h_d_instances = (Instance *)malloc(numElements * sizeof(Instance));
	hipMemcpy(h_d_instances, d_instances, numElements*sizeof(Instance), hipMemcpyDeviceToHost);
	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		hipFree(h_d_instances[i].attribs);
	}
    hipFree(d_instances);
	free(h_d_instances);
	return predictions;
}

int* basicCudaKNN(ArffData* dataset, int k)
{
    // predictions is the array where you have to return the class predicted (integer) for the dataset instances
    int* predictions = (int*)malloc(dataset->num_instances() * sizeof(int));

    // The following two lines show the syntax to retrieve the attribute values and the class value for a given instance in the dataset
    // float attributeValue = dataset->get_instance(instanceIndex)->get(attributeIndex)->operator float();
    // int classValue =  dataset->get_instance(instanceIndex)->get(dataset->num_attributes() - 1)->operator int32();

    // Implement the KNN here, fill the predictions array
	cout << "K is " << k << endl;

	int numElements = dataset->num_instances();
	int numAttribs = dataset->num_attributes() - 1; //-1 because the last attrib is class

	Instance * h_instances = (Instance *)malloc(numElements * sizeof(Instance));

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements / threadsPerBlock) + 1;

	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);


	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		float * attribs = (float *) malloc (sizeof(float)*numAttribs);
		for(int h = 0; h < numAttribs; h++) // compute the distance between the two instances
		{
			attribs[h] = dataset->get_instance(i)->get(h)->operator float();
		}
		h_instances[i].attribs = attribs;
		h_instances[i].cls = dataset->get_instance(i)->get(numAttribs)->operator int32();
	}

    Instance * d_instances;
	hipMalloc(&d_instances, numElements*sizeof(Instance));
	hipMemcpy(d_instances, h_instances, numElements*sizeof(Instance), hipMemcpyHostToDevice);


	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		float * d_attribs;
		hipMalloc(&d_attribs, numAttribs*sizeof(float));
		// Copy up attributes for each instance separately
		hipMemcpy(d_attribs, h_instances[i].attribs, numAttribs*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&(d_instances[i].attribs), &d_attribs, sizeof(float*), hipMemcpyHostToDevice);
	}


	int * d_predictions;
	hipMalloc(&d_predictions, numElements*sizeof(int));

	basicCuda<<<blocksPerGrid, threadsPerBlock>>>(d_instances, numElements,
			numAttribs, k, d_predictions);

	// Copy the device prediction vector in device memory to the host prediction vector
    hipMemcpy(predictions, d_predictions, numElements * sizeof(int), hipMemcpyDeviceToHost);

    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess) {
        fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

    // Free host memory
   	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		free(h_instances[i].attribs);
	}
    free(h_instances);

    // Free device global memory
    hipFree(d_predictions);

	Instance * h_d_instances = (Instance *)malloc(numElements * sizeof(Instance));
	hipMemcpy(h_d_instances, d_instances, numElements*sizeof(Instance), hipMemcpyDeviceToHost);
	for(int i = 0; i < numElements; i++) // for each instance in the dataset
	{
		hipFree(h_d_instances[i].attribs);
	}
    hipFree(d_instances);
	free(h_d_instances);
	return predictions;
}


int* KNN(ArffData* dataset, int k)
{
    // predictions is the array where you have to return the class predicted (integer) for the dataset instances
    int* predictions = (int*)malloc(dataset->num_instances() * sizeof(int));
    
    // The following two lines show the syntax to retrieve the attribute values and the class value for a given instance in the dataset
    // float attributeValue = dataset->get_instance(instanceIndex)->get(attributeIndex)->operator float();
    // int classValue =  dataset->get_instance(instanceIndex)->get(dataset->num_attributes() - 1)->operator int32();
    
    // Implement the KNN here, fill the predictions array
	cout << "K is " << k << endl;
	
	for(int i = 0; i < dataset->num_instances(); i++) // for each instance in the dataset
	{
		//Initialize an empty neighbor
		Neighbor  neighbor;
		neighbor.distance = FLT_MAX;
		neighbor.cls = -1;
		
		//List of k neighbors
		std::list<Neighbor> neighbors (k, neighbor);
	
		for(int j = 0; j < dataset->num_instances(); j++) // target each other instance
		{
			if(i == j) continue;
			
			float distance = 0;
			
			for(int h = 0; h < dataset->num_attributes() - 1; h++) // compute the distance between the two instances
			{
				float diff = dataset->get_instance(i)->get(h)->operator float() - dataset->get_instance(j)->get(h)->operator float();
				distance += diff * diff; 
			}
			
			distance = sqrt(distance);
			
			for (std::list<Neighbor>::iterator it = neighbors.begin(); it != neighbors.end(); it++) {
				if(distance < (*it).distance) {
					Neighbor neighbor;
					neighbor.distance = distance;
					neighbor.cls = dataset->get_instance(j)->get(dataset->num_attributes() - 1)->operator int32();
					neighbors.insert(it, neighbor);
					neighbors.pop_back(); //Remove the last neighbor
					break;
				}
			}
		}
		
		predictions[i] = majorityVote(neighbors);
	}
	
    return predictions;
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset)
{
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matrix size numberClasses x numberClasses
    
    for(int i = 0; i < dataset->num_instances(); i++) // for each instance compare the true class and predicted class
    {
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];
        
        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }
    
    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset)
{
    int successfulPredictions = 0;
    
    for(int i = 0; i < dataset->num_classes(); i++)
    {
        successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagonal are correct predictions
    }
    
    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char *argv[])
{
    if(argc != 3)
    {
        cout << "Usage: ./main <path to dataset> <k>" << endl;
        exit(0);
    }
    
    // Open the dataset
    ArffParser parser(argv[1]);
    ArffData *dataset = parser.parse();
    struct timespec start, end;
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    
    // Get the class predictions
    //int* predictions = KNN(dataset, atoi(argv[2]));
	//int* predictions = basicCudaKNN(dataset, atoi(argv[2]));
	int* predictions = advanceCudaKNN(dataset, atoi(argv[2]));

    // Compute the confusion matrix
    int* confusionMatrix = computeConfusionMatrix(predictions, dataset);
    // Calculate the accuracy
    float accuracy = computeAccuracy(confusionMatrix, dataset);
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;
  
    printf("The KNN classifier for %lu instances required %llu ms CPU time, accuracy was %.4f\n", dataset->num_instances(), (long long unsigned int) diff, accuracy);
	
	//Free the memory
	free(predictions);
	free(confusionMatrix);
}
